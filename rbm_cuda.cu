#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#include "rbm_cuda.cuh"

// #include <iostream>
// using namespace std;

/*
 * Generate a vector of length N with random single-precision floating-point
 * values between 0 and 100.
 */
void generate_random_vector(int N, float **outX)
{
    int i;
    double rMax = (double)RAND_MAX;
    float *X = (float *)malloc(sizeof(float) * N);

    for (i = 0; i < N; i++)
    {
        int r = rand();
        double dr = (double)r;
        X[i] = (dr / rMax) * 100.0;
    }

    *outX = X;
}

/*
 * Generate a matrix with M rows and N columns in column-major order. The matrix
 * will be filled with random single-precision floating-point values between 0
 * and 100.
 */
void generate_random_dense_matrix(int M, int N, float **outA)
{
    int i, j;
    double rMax = (double)RAND_MAX;
    float *A = (float *)malloc(sizeof(float) * M * N);

    // For each column
    for (j = 0; j < N; j++)
    {
        // For each row
        for (i = 0; i < M; i++)
        {
            double dr = (double)rand();
            A[j * M + i] = (dr / rMax) * 100.0;
        }
    }

    *outA = A;
}

void cudaTest() {

	hipblasHandle_t handle = 0;
	// create the cuBLAS handle
	hipblasCreate(&handle);

	int M = 8;
	int N = 8;


	float *A;
	float *X;
	float *Y;
	float *dA;
	float *dX;
	float *dY;

	generate_random_dense_matrix(M, N, &A);
	generate_random_vector(N, &X);
	generate_random_vector(M, &Y);

	hipMalloc((void**) &dA, sizeof(float) * M * N);
	hipMalloc((void**) &dX, sizeof(float) * N);
	hipMalloc((void**) &dY, sizeof(float) * M);

	hipblasSetMatrix(M, N, sizeof(float), A, M, dA, M);


	// // // copy 2th row from A to dX
	// hipblasSetVector(N, sizeof(float), dA+2, M, dX, 1);

	hipblasSetVector(M, sizeof(float), dA + 2 * M, 1, dX, 1);
	// hipblasSetVector(M, sizeof(float), A+2, 1, dX, 1);
	// hipblasSetVector(M, sizeof(float), Y, 1, dY, 1);


	hipMemcpy(X, dX, sizeof(float) * N, hipMemcpyDeviceToHost);



	// for (int j = 0; j < N; j++) {
	// 	for (int i = 0; i < M; i++) {
	// 		cout << A[j*M + i] << " ";
	// 	}
	// 	cout << endl;
	// }

	// for (int j = 0; j < M; j++) {
	// 	cout << X[j] << endl;
	// }



	hipblasDestroy(handle);

}


__global__
void trainKernel(int* users, int* movies, int* ratings, int* starts, int* sizes, 
	float* A, float* B, float* BV, float* BH, int F, int C,
	float* Vzeros, float* Vts, float* Hzeros, float* Hts, float* W_users,
	int batch_size, int CD_K, hipblasHandle_t &handle) {


	// set up cuBlas multiplication parameters
	int ldA = K;
	int ldB = C;
	int ldW = K;
	int ldV = K;
	int ldH = F;

	const float ONE = 1;
	const float ZERO = 0;
	const float* pONE = &ONE;
	const float* pZERO = &ZERO;

	unsigned int index = blockIdx.x * blockDim.x + threadIdx.x;
	while (index < batch_size) {
		// TODO: Write me
		// int user = users[index]; // user id
		int start = starts[index]; // the start index of movies in the batch
		int size = sizes[index]; // number of movies for this user

		float* H0 = Hzeros + index * F; // dim = F * 1
		float* Ht = Hts + index * F; // dim = F * 1

		float* V0 = Vzeros + start * K; // dim = K * size
		float* Vt = Vts + start * K; // dim = K * size

		float* W_user = W_users + start * K * F; // dim = K * F * size


		// from start to (start + size)
		int* uMovies = movies + start; // dim = size
		int* uRatings = ratings + start; // dim = size

		

		// set up V0 and Vt based on the input data.
		for (int i = 0; i < size; i++) {
			V0[i * K + uRatings[i] - 1] = 1;
			Vt[i * K + uRatings[i] - 1] = 1;
			

			// Operation: W_user.slice(i) = A.slice(r.movie) * B;
			// W_user.slice(i) -> K * F
			// A.slice(r.movie) -> K * C
			// B -> C * F

			hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, K, F, C, pONE, A + uMovies[i] * K * C, ldA, B, ldB, pZERO, W_user + i * K * F, ldW);
		}


		/*////////////////// set up H0 by V -> H //////////////////
		H0(j) = sigma( BH(j) + sum_ik ( W(k, j, r.movie) * V0(k, i) ))*/

		// H0 = BH;
		// for (int i = 0; i < size; i++) {

		// 	H0 += W_user.slice(i).t() * V0.col(i);
		// }
		// H0 = 1.0 / (1 + exp(-H0));


		
		/*	W_user.slice(i).t() -> (K * F).t()
			V0.col(i) -> (F * 1)	*/


		// for (int j = 0; j < F; j++) {
		// 	H0[j] = BH[j];
		// }
		hipblasScopy(handle, F, BH, 1, H0, 1);

		// for (int i = 0; i < size; i++) {
		// 	hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, K, F, 1, pONE, W_user + i * K * F, ldW, V0 + i * K, ldV, pONE, H0, ldH);
		// }
		hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, K, F, 1, pONE, (const float**)&W_user, ldW, (const float**)&V0, ldV, pONE, &H0, ldH, size);

		for (int j = 0; j < F; j++) {
			H0[j] = 1.0 / (1 + exp(-H0[j]));
		}


		/*
		/////////////////// Do the contrastive divergence ///////////////////
		for (int n = 0; n < CD_K; n++) {

			////////////// positive phase: V -> H /////////
			Ht = BH;
			for (int i = 0; i < size; i ++) {
				// Ht += W.slice(ims[i]).t() * Vt.col(i);
				Ht += W_user.slice(i).t() * Vt.col(i);
			}
			Ht = 1.0 / (1 + exp(-Ht));
			

			// negative phase: H -> V
			for (int i = 0; i < size; i++) {
				// Vt.col(i) = exp(BV.col(ims[i]) + W.slice(ims[i]) * Ht);
				Vt.col(i) = exp(BV.col(ims[i]) + W_user.slice(i) * Ht);
			}

			// Normalize Vt -> sum_k (Vt(k, i)) = 1
			Vt = normalise(Vt, 1);

		}
		*/

		/////////////////// Do the contrastive divergence ///////////////////
		for (int n = 0; n < CD_K; n++) {
			////////////// positive phase: V -> H /////////
			hipblasScopy(handle, F, BH, 1, Ht, 1);
			hipblasSgemmBatched(handle, HIPBLAS_OP_T, HIPBLAS_OP_N, K, F, 1, pONE, (const float**)&W_user, ldW, (const float**)&Vt, ldV, pONE, &Ht, ldH, size);
			for (int j = 0; j < F; j++) {
				Ht[j] = 1.0 / (1 + exp(-Ht[j]));
			}

			// negative phase: H -> V
		}


		index += blockDim.x * gridDim.x;
	}
}


void train(int* users, int* movies, int* ratings, int* starts, int* sizes, 
	float* A, float* B, float* BV, float* BH, int F, int C,
	float* Vzeros, float* Vts, float* Hzeros, float* Hts, float* W_users,
	int batch_size, int CD_K) {

	int block_size = (batch_size < 512) ? batch_size : 512;
	int grid_size = (batch_size + block_size -1) / block_size;


	// create the cuBLAS handle
	hipblasHandle_t handle = 0;
	hipblasCreate(&handle);

	trainKernel<<<grid_size, block_size>>>(users, movies, ratings, starts, sizes, 
		A, B, BV, BH, F, C, Vzeros, Vts, Hzeros, Hts, W_users, batch_size, CD_K, handle); 

	hipblasDestroy(handle);
}
























